/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
    

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
    return; 
}


void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 256; 

    //INSERT CODE HERE
    dim3 dimGrid((n - 1) / BLOCK_SIZE + 1, 1, 1);
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    VecAdd <<<dimGrid, dimBlock>>> (n, A, B, C); 
    return ;
}

